#include "hip/hip_runtime.h"
#include "cuda_wrapper.h"
#include "gmem_alloc.h"

#include "errchk.cuh"
#include "utils_CUDA.cuh"
#include "macro.h"
#include "timer.h"
#include "myhelper.cuh"

#include "bwa.h"
#include "bwt_CUDA.cuh"
#include "bntseq.h"
#include "fastmap.h"

#include "hashKMerIndex.h"
#include "seed.cuh"
#include "aux.cuh"

#include <string.h>
#include <fstream>
#include <iostream>

#include <hipcub/hipcub.hpp>
#include <thrust/device_vector.h>
#include <thrust/sort.h>


#define PRINT(LABEL) \
	g3_opt->print_mask & BIT(LABEL)

#define CUDA_TIMER_INIT \
	hipEvent_t timer_event_start, timer_event_stop;\
	CUDA_CHECK(hipEventCreate(&timer_event_start));\
	CUDA_CHECK(hipEventCreate(&timer_event_stop));

#define CUDA_TIMER_DESTROY \
	CUDA_CHECK(hipEventDestroy(timer_event_start));\
	CUDA_CHECK(hipEventDestroy(timer_event_stop));

#define CUDA_TIMER_START(lap) \
	lap = 0;\
	CUDA_CHECK(hipEventRecord(timer_event_start, 0));

//CUDA_CHECK(hipEventRecord(timer_event_start, 0));\

#define CUDA_TIMER_END(lap) \
	CUDA_CHECK(hipEventRecord(timer_event_stop, 0));\
	CUDA_CHECK(hipEventSynchronize(timer_event_stop));\
	CUDA_CHECK(hipEventElapsedTime(&lap, timer_event_start, timer_event_stop));\

#define CUDA_CHECK_KERNEL_RUN()\
{\
	hipError_t err;\
	err = hipGetLastError();\
	if(err != hipSuccess)\
	{\
		fprintf(stderr,"GPU %d hipGetLastError(): %s %s %d\n", gpuid, hipGetErrorString(err), __FILE__, __LINE__);\
	}\
	err = hipDeviceSynchronize();\
	if(err != hipSuccess)\
	{\
		fprintf(stderr,"GPU %d hipDeviceSynchronize(): %s %s %d\n", gpuid, hipGetErrorString(err), __FILE__, __LINE__);\
		/*CUDA_CHECK(hipDeviceReset());\*/\
		/*while((err = hipGetLastError()) != hipSuccess) {;} \*/\
		return 1;\
	}\
}


extern float tprof[MAX_NUM_GPUS][MAX_NUM_STEPS];

/*
 *
 * Stage    |   Substage        |   Step          |   note
 * ---------------------------------------------------------------------------
 * Seeding  |   SMEM seeding    |   seed          |   seeding core
 *          |   Reseeding       |   r2            |
 *          |                   |   r3            |
 * Chaining |   B-tree chaining |   sal           |
 *          |                   |   sort_seeds    |
 *          |                   |   chain         |   chaining core
 *          |                   |   sort_chains   |
 *          |                   |   filter        |
 * Extending|   Local extending |   pairgen       |
 *          |                   |   extend        |   extending core
 *          |                   |   filter_mark   |
 *          |                   |   sort_alns     |
 *          |   Traceback       |   pairgen       |
 *          |                   |   traceback     |   traceback core (w/ NM test)
 *          |                   |   finalize      |
 */
int bwa_align(int gpuid, process_data_t *proc, g3_opt_t *g3_opt)
{
	float step_lap;
	void *d_temp_storage;
	size_t temp_storage_size;
	int batch_size, num_seeds_to_extend;
	int batch_num_alns;
	if((batch_size = proc->batch_size) == 0){
		return 0;
	}
	CUDA_TIMER_INIT;
	// (1/3) Seeding
	// SMEM seeding (Seeding 1/2)
	CUDA_TIMER_START(step_lap);
	preseedAndFilterV2 <<< batch_size, 320, 0, 0 >>> (
			proc->d_fmIndex,
			proc->d_opt, proc->d_seq, proc->d_seq_offset,
			proc->d_aux, proc->d_kmerHashTab, proc->d_buffer_pools);
	CUDA_CHECK_KERNEL_RUN();
	CUDA_TIMER_END(step_lap);
	tprof[gpuid][S_SMEM] += step_lap;
	if(PRINT(_SMEM) || PRINT(_ALL_SEEDING)){
		for(int readID = 0; readID < batch_size; readID++)
		{
			printIntv<<<1, WARPSIZE, 0, 0>>>(proc->d_aux, readID, _SMEM);
			CUDA_CHECK_KERNEL_RUN();
		}
	}
	// Reseeding (Seeding 2/2)
	CUDA_TIMER_START(step_lap);
	reseedV2 <<< ceil((float)batch_size / WARPSIZE) == 0 ? 1 
		: ceil((float)batch_size / WARPSIZE), WARPSIZE, 0, 0 >>>(
				proc->d_fmIndex, proc->d_opt, proc->d_seq, proc->d_seq_offset,
				proc->d_aux, proc->d_kmerHashTab, 
				proc->d_buffer_pools, batch_size);
	CUDA_CHECK_KERNEL_RUN();
	CUDA_TIMER_END(step_lap);
	tprof[gpuid][S_R2] += step_lap;
	// Reseeding 3rd Round
	CUDA_TIMER_START(step_lap);
	reseedLastRound <<< ceil((float)batch_size / WARPSIZE) == 0 ? 1 
		: ceil((float)batch_size / WARPSIZE), WARPSIZE, 0, 0 >>>(
				proc->d_fmIndex, proc->d_opt, proc->d_seq,
			       	proc->d_seq_offset,
				proc->d_aux, proc->d_kmerHashTab, batch_size);
	CUDA_CHECK_KERNEL_RUN();
	CUDA_TIMER_END(step_lap);
	tprof[gpuid][S_R3] += step_lap;
	if(PRINT(_INTV) || PRINT(_ALL_SEEDING)){
		for(int readID = 0; readID < batch_size; readID++)
		{
			printIntv<<<1, WARPSIZE, 0, 0>>>(proc->d_aux, readID, _INTV);
			CUDA_CHECK_KERNEL_RUN();
		}
	}
	// (2/3) Chaining
	// B-tree chaining (Chaining 1/1)
	CUDA_TIMER_START(step_lap);
	saLookup <<< batch_size, 128, 0, 0 >>> (
			proc->d_opt, proc->d_bwt, proc->d_bns,
		       	proc->d_seq, proc->d_seq_offset, proc->d_aux,
			proc->d_seq_seeds,
			proc->d_buffer_pools);
	CUDA_CHECK_KERNEL_RUN();
	CUDA_TIMER_END(step_lap);
	tprof[gpuid][C_SAL] += step_lap;
	if(PRINT(_SEED) || PRINT(_ALL_CHAINING)){
		for(int readID = 0; readID < batch_size; readID++)
		{
			printSeed<<<1, WARPSIZE, 0, 0>>>(proc->d_seq_seeds, readID, _SEED);
			CUDA_CHECK_KERNEL_RUN();
		}
	}
	CUDA_TIMER_START(step_lap);
	// sort translated seeds in rbeg order
	sortSeedsLowDim 
		<<< batch_size, SORTSEEDSLOW_BLOCKDIMX, 0, 0 >>> (
				proc->d_seq_seeds,
				proc->d_buffer_pools);
	CUDA_CHECK_KERNEL_RUN();
	sortSeedsHighDim 
		<<< batch_size, SORTSEEDSHIGH_BLOCKDIMX, 0, 0 >>> (
				proc->d_seq_seeds,
				proc->d_buffer_pools);
	CUDA_CHECK_KERNEL_RUN();
	CUDA_TIMER_END(step_lap);
	tprof[gpuid][C_SORT_SEEDS] += step_lap;
	if(PRINT(_STSEED) || PRINT(_ALL_CHAINING)){
		for(int readID = 0; readID < batch_size; readID++)
		{
			printSeed<<<1, WARPSIZE, 0, 0>>>(proc->d_seq_seeds, readID, _STSEED);
			CUDA_CHECK_KERNEL_RUN();
		}
	}
	CUDA_TIMER_START(step_lap);
#if 0
#define SEEDCHAINING_CHAIN_BLOCKDIMX 256
	SEEDCHAINING_chain_kernel <<< batch_size, SEEDCHAINING_CHAIN_BLOCKDIMX, 0, 0 >>> (
			proc->d_opt, proc->d_bns, proc->d_seq, proc->d_seq_offset, proc->d_seq_seeds,
			proc->d_chains,	// output
			proc->d_buffer_pools);
#else
	BTreeChaining 
		<<< ceil((float)batch_size / WARPSIZE) == 0 ? 1 : ceil((float)batch_size / WARPSIZE), WARPSIZE, 0, 0 >>>(
				batch_size,
				proc->d_opt, proc->d_bns,
			       	proc->d_seq, proc->d_seq_offset, proc->d_seq_seeds,
				proc->d_chains,	// output
				proc->d_buffer_pools);
#endif
	CUDA_CHECK_KERNEL_RUN();
	CUDA_TIMER_END(step_lap);
	tprof[gpuid][C_CHAIN] += step_lap;
	if(PRINT(_CHAIN) || PRINT(_ALL_CHAINING)){
		for(int readID = 0; readID < batch_size; readID++){
			printChain<<<1, WARPSIZE, 0, 0>>>(proc->d_chains, readID, _CHAIN);
			CUDA_CHECK_KERNEL_RUN();
		}
	}
	// sort before filtering
	CUDA_TIMER_START(step_lap);
	sortChainsDecreasingWeight 
		<<< batch_size, SORTCHAIN_BLOCKDIMX, 
		MAX_N_CHAIN*2*sizeof(uint16_t)+sizeof(mem_chain_t**), 0>>> 
			(proc->d_chains, proc->d_buffer_pools);
	CUDA_CHECK_KERNEL_RUN();
	CUDA_TIMER_END(step_lap);
	tprof[gpuid][C_SORT_CHAINS] += step_lap;
	if(PRINT(_STCHAIN) || PRINT(_ALL_CHAINING)){
		for(int readID = 0; readID < batch_size; readID++)
		{
			printChain<<<1, WARPSIZE, 0, 0>>>(proc->d_chains, readID, _STCHAIN);
			CUDA_CHECK_KERNEL_RUN();
		}
	}
	// filter chains
	CUDA_TIMER_START(step_lap);
	CHAINFILTERING_filter_kernel <<< batch_size, CHAIN_FLT_BLOCKSIZE, MAX_N_CHAIN*(3*sizeof(uint16_t)+sizeof(uint8_t)), 0 >>> (
			proc->d_opt, 
			proc->d_chains, 	// input and output
			proc->d_buffer_pools);
	CUDA_CHECK_KERNEL_RUN();
	CUDA_TIMER_END(step_lap);
	tprof[gpuid][C_FILTER] += step_lap;
	if(PRINT(_FTCHAIN) || PRINT(_ALL_CHAINING)){
		for(int readID = 0; readID < batch_size; readID++)
		{
			printChain<<<1, WARPSIZE, 0, 0>>>(proc->d_chains, readID, _FTCHAIN);
			CUDA_CHECK_KERNEL_RUN();
		}
	}
	// (3/3) Extending
	// Extending -> Local extending (1/2)
	CUDA_TIMER_START(step_lap);
	SWSeed <<< ceil((float)batch_size / WARPSIZE) == 0 ? 1 : ceil((float)batch_size / WARPSIZE), WARPSIZE, 0, 0 >>>(
			proc->d_chains, proc->d_regs,
		       	proc->d_seed_records, proc->d_Nseeds, batch_size,
			proc->d_buffer_pools
			);
	CUDA_CHECK_KERNEL_RUN();
	CUDA_CHECK(hipMemcpy(&num_seeds_to_extend, proc->d_Nseeds, sizeof(int), hipMemcpyDeviceToHost));

	if(num_seeds_to_extend==0){
		proc->n_processed += proc->batch_size;
		return 0;
	}
	// pair gen
	ExtendingPairGenerate <<< ceil((float)num_seeds_to_extend/32.0), 32, 0, 0 >>> (
			proc->d_opt, proc->d_bns, proc->d_pac, proc->d_seq, proc->d_seq_offset,
			proc->d_chains, proc->d_regs, proc->d_seed_records, proc->d_Nseeds,
			batch_size, proc->d_buffer_pools
			);
	CUDA_CHECK_KERNEL_RUN();
	CUDA_TIMER_END(step_lap);
	tprof[gpuid][E_PAIRGEN] += step_lap;
	if(PRINT(_DETAIL)){
		std::cerr << "GPU " << gpuid << "  " << "# local extending pairs: " << num_seeds_to_extend << std::endl;
	}
	if(PRINT(_EXPAIR) || PRINT(_ALL_EXTENDING)){
		printPair<<<1, WARPSIZE, 0, 0>>>(proc->d_seed_records, num_seeds_to_extend, _EXPAIR);
		CUDA_CHECK_KERNEL_RUN();
	}
	// local ext (Smith-Waterman)
	CUDA_TIMER_START(step_lap);
	localExtending <<< num_seeds_to_extend, WARPSIZE, 0, 0 >>> (
			proc->d_opt,
			proc->d_chains, 		// input chains
			proc->d_seed_records,
			proc->d_regs,		// output array
			proc->d_Nseeds);
	CUDA_CHECK_KERNEL_RUN();
	CUDA_TIMER_END(step_lap);
	tprof[gpuid][E_EXTEND] += step_lap;
	if(PRINT(_REGION) || PRINT(_ALL_EXTENDING)){
		for(int readID = 0; readID < batch_size; readID++)
		{
			printReg<<<1, WARPSIZE, 0, 0>>>(proc->d_regs, readID, _REGION);
			CUDA_CHECK_KERNEL_RUN();
		}
	}
	// filter regions (which are extension results)
	CUDA_TIMER_START(step_lap);
	filterRegions <<< batch_size, 320, 0, 0 >>> (
			proc->d_opt, proc->d_bns,
			proc->d_chains, proc->d_regs, proc->d_buffer_pools
			);
	CUDA_CHECK_KERNEL_RUN();
	CUDA_TIMER_END(step_lap);
	tprof[gpuid][E_FILTER_MARK] += step_lap;
	if(PRINT(_FTREGION) || PRINT(_ALL_EXTENDING)){
		for(int readID = 0; readID < batch_size; readID++)
		{
			printReg<<<1, WARPSIZE, 0, 0>>>(proc->d_regs, readID, _FTREGION);
			CUDA_CHECK_KERNEL_RUN();
		}
	}
	// sort regions before traceback
	CUDA_TIMER_START(step_lap);
	sortRegions <<< ceil((float)batch_size / WARPSIZE) == 0 ? 1 : ceil((float)batch_size / WARPSIZE), WARPSIZE, 0, 0 >>>(
			proc->d_regs, batch_size, proc->d_buffer_pools);
	CUDA_CHECK_KERNEL_RUN();
	CUDA_TIMER_END(step_lap);
	tprof[gpuid][E_SORT_ALNS] += step_lap;
	if(PRINT(_STREGION) || PRINT(_ALL_EXTENDING)){
		for(int readID = 0; readID < batch_size; readID++)
		{
			printReg<<<1, WARPSIZE, 0, 0>>>(proc->d_regs, readID, _STREGION);
			CUDA_CHECK_KERNEL_RUN();
		}
	}
	// TODO  cleanup these
	// compute offsets to the result alignments array.
	compute_offsets_on_host(proc, batch_size);

	// launch verifier
	// verify_scan_kernel<<<1,1>>>(proc->d_regs, proc->d_offsets, batch_size);
	// hipDeviceSynchronize();

	// Extending -> Traceback (2/2)
	CUDA_TIMER_START(step_lap);
	CUDA_CHECK(hipMemset(proc->d_Nseeds, 0, sizeof(int)));
	// preproc
	FINALIZEALN_preprocessing1_kernel <<< ceil((float)batch_size / WARPSIZE) == 0 ? 1 : ceil((float)batch_size / WARPSIZE), WARPSIZE, 0, 0 >>>(
			batch_size,
			proc->d_regs, proc->d_alns, proc->d_seed_records, proc->d_Nseeds,
			proc->d_buffer_pools);
	CUDA_CHECK_KERNEL_RUN();

	CUDA_CHECK(hipMemcpy(&batch_num_alns, proc->d_Nseeds, sizeof(int), hipMemcpyDeviceToHost));
	if(batch_num_alns==0){
		proc->n_processed += proc->batch_size;
		return 0;
	}
	// preproc
	FINALIZEALN_preprocessing2_kernel <<< ceil((float)batch_num_alns / WARPSIZE) == 0 ? 1 : ceil((float)batch_num_alns / WARPSIZE), WARPSIZE, 0, 0 >>>(
			proc->d_opt, proc->d_seq, proc->d_seq_offset,
			proc->d_pac, proc->d_bns,
			proc->d_regs, proc->d_alns, proc->d_seed_records, batch_num_alns,
			proc->d_sortkeys_in,	// sortkeys_in = bandwidth * rlen
			proc->d_seqIDs_in,
			proc->d_buffer_pools);
	CUDA_CHECK_KERNEL_RUN();
	// preproc: reverse query and target if aln position is on reverse strand
	FINALIZEALN_reverseSeq_kernel <<< batch_num_alns, WARPSIZE, 0, 0 >>> (proc->d_seed_records, proc->d_alns, proc->d_buffer_pools);
	CUDA_CHECK_KERNEL_RUN();
	// preproc: sort
	// determine temporary storage requirement
	d_temp_storage = NULL;
	temp_storage_size = 0;
	CUDA_CHECK(hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_size, proc->d_sortkeys_in, proc->d_sortkeys_out, proc->d_seqIDs_in, proc->d_seqIDs_out, batch_num_alns, 0, 8*sizeof(int), 0));
	// Allocate temporary storage
	CUDA_CHECK(hipMalloc(&d_temp_storage, temp_storage_size));
	// perform radix sort
	CUDA_CHECK(hipcub::DeviceRadixSort::SortPairsDescending(d_temp_storage, temp_storage_size, proc->d_sortkeys_in, proc->d_sortkeys_out, proc->d_seqIDs_in, proc->d_seqIDs_out, batch_num_alns, 0, 8*sizeof(int), 0));
	hipFree(d_temp_storage);
	CUDA_TIMER_END(step_lap);
	tprof[gpuid][E_T_PAIRGEN] += step_lap;
	if(PRINT(_TBPAIR) || PRINT(_ALL_EXTENDING)){
		printPair<<<1, WARPSIZE, 0, 0>>>(proc->d_seed_records, batch_num_alns, _TBPAIR);
		CUDA_CHECK_KERNEL_RUN();
	}

	if(PRINT(_DETAIL)){
		std::cerr << "GPU " << gpuid << "  " << "# traceback pairs: " << batch_num_alns << std::endl;
	}
	// traceback (with Needlman-Wunsch test)
	CUDA_TIMER_START(step_lap);
	traceback<<< ceil((float)batch_num_alns / WARPSIZE) == 0 ? 1 : ceil((float)batch_num_alns / WARPSIZE), WARPSIZE, 0, 0 >>>(
			proc->d_opt,
			proc->d_seed_records, batch_num_alns, proc->d_alns, proc->d_seqIDs_out,
			proc->d_buffer_pools
			);
	CUDA_CHECK_KERNEL_RUN();
	CUDA_TIMER_END(step_lap);
	tprof[gpuid][E_TRACEBACK] += step_lap;
	if(PRINT(_ALIGNMENT) || PRINT(_ALL_EXTENDING)){
		for(int readID = 0; readID < batch_size; readID++)
		{
			printAln<<<1, WARPSIZE, 0, 0>>>(proc->d_bns, proc->d_alns, readID, _ALIGNMENT);
			CUDA_CHECK_KERNEL_RUN();
		}
	}
	// pack to memcpy to host
	CUDA_TIMER_START(step_lap);
	finalize<<< ceil((float)batch_num_alns / WARPSIZE) == 0 ? 1 : ceil((float)batch_num_alns / WARPSIZE), WARPSIZE, 0, 0 >>>(
			proc->d_opt, proc->d_bns, proc->d_seq, proc->d_seq_offset,
			proc->d_regs, proc->d_alns, proc->d_seed_records, batch_num_alns,
			proc->d_buffer_pools,
			proc->d_offsets,
			proc->d_rids,
			proc->d_positions,
			proc->d_ncigars,
			proc->d_cigars
			);
	CUDA_CHECK_KERNEL_RUN();
	CUDA_TIMER_END(step_lap);
	tprof[gpuid][E_FINALIZE] += step_lap;
	if(PRINT(_RESULT) || PRINT(_ALL_EXTENDING)){
		for(int readID = 0; readID < batch_size; readID++)
		{
			printAln<<<1, WARPSIZE, 0, 0>>>(proc->d_bns, proc->d_alns, readID, _RESULT);
			CUDA_CHECK_KERNEL_RUN();
		}
	}
	// TODO cleanup these
	// verify if results are correctly written into the arrays
	// verify_writing_kernel<<<1,1>>>(proc->d_offsets, batch_size,
	// proc->d_rids, proc->d_positions, proc->d_ncigars, proc->d_cigars);
	// hipDeviceSynchronize();

	//printBufferInfoHost(proc->d_buffer_pools);
	CUDA_TIMER_DESTROY;
	return 0;
}
